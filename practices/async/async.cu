
#include <hip/hip_runtime.h>
#include <iostream>
#include <future>

__global__
void global_is_prime(int x, bool* result) {
  for (int i=2; i<x; ++i) {
    if (x%i==0) {
      *result = false;
      break;
    }
  }
  *result = true;
}

// a non-optimized way of checking for prime numbers:
bool is_prime (int x) {
  std::cout << "Calculating. Please, wait...\n";
  for (int i=2; i<x; ++i) {
    if (x%i==0) {
      return false;
    }
  }
  return true;
}

int main(int argc, char *argv[]) { 

  const int LargeNumber = 313222313;
  const int SmallNumber = 3;
  bool *devResult, hostResult; 

  /* Allocate memory on device */ 
  hipMalloc((void **)&devResult, sizeof(bool)); 

  global_is_prime<<<1,1>>>(LargeNumber, devResult);
  std::future<bool> future = std::async(std::launch::async, is_prime, SmallNumber);
  std::cout << "Execute is_prime function" << std::endl;

  /* Copy device memory to host */ 
  hipMemcpy(&hostResult, devResult, sizeof(bool), hipMemcpyDeviceToHost); 
  std::cout << "After Memcpy" << std::endl;

  std::future_status status;

  std::cout << "waiting...\n";
  do {
    status = future.wait_for(std::chrono::milliseconds(1));
  }while (status != std::future_status::ready); 

  bool result = future.get();
  std::cout << "Result : " << result << std::endl;

  /* Cleanup */ 
  hipFree(devResult); 

  return 0;
}
